#include "hip/hip_runtime.h"
/*******************************************************************************
*
*  kernelSmoother
*
*  This provides a CUDA implementation of a kernel smooother.
*   http://en.wikipedia.org/wiki/Kernel_smoother
*  The particular kernel in this file is a nearest neighbor smoother
*  in order to keep the code as simple to understand as possible.
*
*  This is implemeneted for 2-d square grids.
*
*  Parameters of note are all in struct CUDAGrid.
*    gridWidth -- size of the grid is gridWidth^2
*    kernelWidth -- region around point x,y to smooth
*        must be odd, i.e. 2k+1 smooths box with corners (x-k,y-k) to (x+k,y+k)
*    blockWidth -- number of processors per block.
*        must be ((cg.gridWidth-(cg.kernelWidth-1))^2 % (blockWidth^2)) == 0 
*        i.e. the smoothed regions must be of blocksize increments.
*
*  The smoothed region is only defined for the interior that has the kernel
*   defined inside the boundary, e.g. for gridWidth=10, kernelWidth=2 the
*   region from 2,2 to 7,7 will be smoothed. 
*
********************************************************************************/

/*******************************************************************************
*
*  CUDA concepts
*
*  This file shows how to use many features of CUDA:
*     2d grids
*     pitch allocation
*
********************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>

//
// CUDAGrid: structure to define geometry parameter.
//   set one of these up in main()
//
typedef struct
{
  unsigned gridWidth;
  unsigned kernelWidth;
  unsigned blockWidth;
} CUDAGrid;

/*------------------------------------------------------------------------------
* Name: NNSmoothKernel
* Action:  The CUDA kernel that implements kernel smoothing.
*             Yuck, that's two senses of kernel.
*-----------------------------------------------------------------------------*/
__global__ void NNSmoothKernel ( float* pFieldIn, float* pFieldOut, size_t pitch, CUDAGrid cg )
{ 
} 


/*------------------------------------------------------------------------------
* Name:  SmoothField
* Action:  Host entry point to kernel smoother
*-----------------------------------------------------------------------------*/
bool SmoothField ( float* pHostFieldIn, float *pHostFieldOut, CUDAGrid cg ) 
{ 
  float * pDeviceFieldIn = 0;
  float * pDeviceFieldOut = 0;

  size_t pitch, pitchout;

  struct timeval ta, tb, tc, td;

  // Check the grid dimensions and extract parameters.  See top description about restrictions
  assert ((( cg.kernelWidth -1 )%2) == 0 );     // Width is odd
  unsigned blockSize = cg.blockWidth * cg.blockWidth;  
  assert( ((cg.gridWidth-(cg.kernelWidth-1))*(cg.gridWidth-(cg.kernelWidth-1)) % blockSize) == 0 );

  gettimeofday ( &ta, NULL );

  // Place the data set on device memory
  hipMallocPitch((void**)&pDeviceFieldIn, &pitch, cg.gridWidth*sizeof(float), cg.gridWidth ); 
  hipMemcpy2D ( pDeviceFieldIn, pitch,
                 pHostFieldIn, cg.gridWidth*sizeof(float), cg.gridWidth*sizeof(float), cg.gridWidth,
                 hipMemcpyHostToDevice); 

  // Allocate the output
  hipMallocPitch((void**)&pDeviceFieldOut, &pitchout, cg.gridWidth*sizeof(float), cg.gridWidth ); 

  gettimeofday ( &tb, NULL );

  // Construct a 2d grid/block
  const dim3 DimBlock .....TODO
  const dim3 DimGrid .....TODO

  // Invoke the kernel
  NNSmoothKernel <<<DimGrid,DimBlock>>> ( pDeviceFieldIn, pDeviceFieldOut, pitch, cg ); 

  gettimeofday ( &tc, NULL );

  // Retrieve the results
  hipMemcpy2D(pHostFieldOut, cg.gridWidth*sizeof(float), 
               pDeviceFieldOut, pitch, cg.gridWidth*sizeof(float), cg.gridWidth,
               hipMemcpyDeviceToHost); 

  gettimeofday ( &td, NULL );


  if ( ta.tv_usec < td.tv_usec )
  {
    printf ("Elapsed total time (s/m): %d:%d\n", td.tv_sec - ta.tv_sec, td.tv_usec - ta.tv_usec );
  } else {
    printf ("Elapsed total time (s/m): %d:%d\n", td.tv_sec - ta.tv_sec - 1, 1000000 - td.tv_usec + ta.tv_usec );
  }

  if ( tb.tv_usec < tc.tv_usec )
  {
    printf ("Elapsed kernel time (s/m): %d:%d\n", tc.tv_sec - tb.tv_sec, tc.tv_usec - tb.tv_usec );
  } else {
    printf ("Elapsed kernel time (s/m): %d:%d\n", tc.tv_sec - tb.tv_sec - 1, 1000000 - tc.tv_usec + tb.tv_usec );
  }

  return true;
}



/*------------------------------------------------------------------------------
* Name:  initField
* Action:  Initialize a field to predictable values.
*    This is a useful format for debugging, because values 
*    accumulate to their initial value.
*-----------------------------------------------------------------------------*/
void initField ( unsigned dim, float* pField )
{
  for ( unsigned j=0; j<dim; j++ )
  {
    for ( unsigned i=0; i<dim; i++ )
    {
      pField[j*dim+i] = j + i;
    }
  }
}


/*------------------------------------------------------------------------------
* Name:  main
* Action:  Entry point
*-----------------------------------------------------------------------------*/
int main ()
{

  // Define the grid
  CUDAGrid cg;
  cg.gridWidth = 4112;
  cg.kernelWidth = 17;
  cg.blockWidth = 16;

  // Create the input field
  float *field = (float *) malloc ( cg.gridWidth * cg.gridWidth * sizeof(float));
  initField ( cg.gridWidth, field );

  // Create the output field
  float *out = (float *) malloc ( cg.gridWidth * cg.gridWidth * sizeof(float));

  // Call the kernel
  SmoothField ( field, out, cg );

  // Print the output field (for debugging purposes.
/*  unsigned koffset = (cg.kernelWidth-1)/2;
  for ( unsigned j=0; j< cg.gridWidth; j++ )
  {
    for ( unsigned i=0; i< cg.gridWidth; i++ )
    {
      if ( ( i >= koffset ) && 
           ( j >= koffset ) &&
           ( i < ( cg.gridWidth - koffset )) &&
           ( j < ( cg.gridWidth - koffset )) )
      {
        printf ("%4.0f, ", out[j*cg.gridWidth + i]);
      }
      else
      {
        printf ("  na, ");
      }
    }  
    printf ("\n");
  }
*/
}

